
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void VecAdd(float* A, float* B, float* C) {
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main() {
    int N = 5;
    size_t size = N * sizeof(float);

    float* A = new float[N];
    float* B = new float[N];
    float* C = new float[N];
    for (int i = 0; i < N; i++) { // this allocates the data on the CPU!
        A[i] = i * 2;
        B[i] = i;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size); // this allocates the data on the GPU!
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice); // host to device == CPU->GPU
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    VecAdd<<<1, N>>>(d_A, d_B, d_C);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost); // host to device == GPU->CPU

    for (int i = 0; i < N; i++) {
        cout << C[i] << endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}